
#include <hip/hip_runtime.h>
#include <stdio.h>

// A simple NxN row-major matrix multiplication
// extern "C" for name unmangling
extern "C" __global__
void matMulKernel(const float* A, const float* B, float* C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float val = 0.0f;
        for (int k = 0; k < N; k++) {
            val += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = val;
    }
}
